#include "hip/hip_runtime.h"
/* randist/multinormal.c
 * 
 * Random Number Distributions - Binomial Distribution
 *
 */

#include <math.h>
#include "randist_cuda.h"
#include "specfunc/sf_gamma_cuda.h"
 
/* The binomial distribution has the form,

   prob(k) =  n!/(k!(n-k)!) *  p^k (1-p)^(n-k) for k = 0, 1, ..., n

   This is the algorithm from Knuth */

__device__
float
ran_binomial_pdf (const unsigned int k, const float p,
                      const unsigned int n)
{
  if (k > n)
    {
      return 0;
    }
  else
    {
      float P;

      if (p == 0) 
        {
          P = (k == 0) ? 1 : 0;
        }
      else if (p == 1)
        {
          P = (k == n) ? 1 : 0;
        }
      else
        {
          float ln_Cnk = sf_lnchoose (n, k);
          P = ln_Cnk + k * logf (p) + (n - k) * log1pf (-p);
          P = expf (P);
        }

      return P;
    }
}
