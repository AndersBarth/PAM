#include "hip/hip_runtime.h"
/* eval_prob_3c_bg.cu - Evaluate probabilty for 3-Color PDA after 
 *       blue and green excitation using CUDA
 *
 *========================================================*/

 /* For Matlab interaction */
#include "mex.h"

 /* Standard c libs */
#include <stdio.h>                       
#include <stdlib.h>
// #include <stddef.h>
// #include <limits.h>
#include <string.h>
#include <errno.h>
//#include <math.h>

//#include "randist/randist_cuda.h"                        /* Statistical distributions */
#include "randist/sys/minmax.h"                            /* Minimum/Maximum Makro */
#include "sys/tcpda_cuda_lib.h"                                /* Data Structure of Calculations */

#include "randist/specfunc/eval_cuda.h"                    /* Data Structure of Calculations */
#include "randist/specfunc/result_cuda.h"                  /* Evaluvating results of calculations */

 /* For gpu calculation using CUDA */ 
#include <hip/hip_runtime.h>

/* Here we define an inline macro for error checking the results of the CUDA APIs
 * It simply saves typing, simply use it with 'check_cuda_errors( <<CUDA_API>> );' */ 
#define check_cuda_errors(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

 /* Forward declarations:
  * I couldn't split the source file without compile errors, even after customizing
  * mex_CUDA_win64.xml (contains matlab (mex) and cuda compile instructions for the
  * nvcc, the nvidia C/C++ Compiler) and adding '-rdc=true' as compile option and the
  * libraries 'cudart.lib' and 'cudadevrt.lib' */

__global__ void eval_prob_3c_bg(tcpda_data *data);
__device__ int sf_lnfact_e(const unsigned int n, sf_result * result);
__device__ int sf_lnchoose_e(unsigned int n, unsigned int m, sf_result * result);
__device__ float sf_lnfact(const unsigned int n);
__device__ float sf_lnchoose(unsigned int n, unsigned int m);
__device__ float ran_multinomial_lnpdf (const size_t K, const float p[], const unsigned int n[]);
__device__ float ran_multinomial_pdf (const size_t K, const float p[], const unsigned int n[]);
__device__ float ran_binomial_pdf (const unsigned int k, const float p, const unsigned int n);

/* The gateway function to matlab */
void mexFunction( int nlhs, mxArray *plhs[],
                  int nrhs, const mxArray *prhs[])
{

    /* check for proper number of arguments */
    if(nrhs!=20) {
        mexErrMsgIdAndTxt("tcPDA:eval_prob_3c_bg:nrhs","20 inputs required.");
    }
    if(nlhs!=1) {
        mexErrMsgIdAndTxt("tcPDA:eval_prob_3c_bg:nlhs","One output required.");
    }

    size_t burst_cols, burst_rows;              /* Dimension of Burst Arrray */
    size_t p_cols, p_rows;                      /* Dimension of Probability Array */
    
    
    /* get dimensions of the burst matrices */
    burst_cols = mxGetN(prhs[0]);
    burst_rows = mxGetM(prhs[0]);
    
    /* get dimensions of the P matrix */
    p_cols = mxGetN(prhs[15]);
    p_rows = mxGetM(prhs[15]);

    /* The burst_row size (column length) determines how many warps are required for 
     * calculation to dynamically maximize occupancy (i.e. utilization).
     * See the CUDA Programming Guide for C and the CUDA_Occupancy_calculator.xls together
     * with the output of nvcc with the options '--resource-usage' and '-Xptxas=-v' for more */
    const int warp_size = 32;				/* By definition 32 for CUDA enabled devices, 
											 * so we always start a multiple of 32 threads */
    const int max_grid_size = 112; 			/* this is 8 blocks (112 / 32) per MP for a compute
											 * capability 5.2 device. (Maxwell GPU (GTX 9x0)) */

    int warp_count = (burst_rows / warp_size) + (((burst_rows % warp_size) == 0) ? 0 : 1);
    int warp_per_block = max(1, min(4, warp_count));

    /* For the eval_prob_3c_bg kernel, the block size is allowed to grow to
     * four warps per block, and the block count becomes the warp count over four
     * or the GPU "fill" whichever is smaller */
    int thread_count = warp_size * warp_per_block;
    int block_count = min( max_grid_size, max(1, warp_count/warp_per_block) );
    dim3 block_dim = dim3(thread_count, 1, 1);
    dim3 grid_dim  = dim3(block_count, 1, 1);

    /* Sizes for gpu memory allocation */
    size_t size_burst_matrix, size_p_matrix, size_lib_b, size_lib_t;

    size_burst_matrix = burst_cols * burst_rows * sizeof(float);
    size_p_matrix = p_cols * p_rows * sizeof(float);
    size_t vNBGbb, vNBGbg, vNBGbr, vNBGgg, vNBGgr;
    vNBGbb = mxGetScalar(prhs[5]);
    vNBGbg = mxGetScalar(prhs[6]);
    vNBGbr = mxGetScalar(prhs[7]);
    vNBGgg = mxGetScalar(prhs[8]);
    vNBGgr = mxGetScalar(prhs[9]);
    size_lib_b = burst_cols * burst_rows * (vNBGgg+1) * (vNBGgr+1) * sizeof(float);
    size_lib_t = burst_cols * burst_rows * (vNBGbb+1) * (vNBGbg+1) * (vNBGbr+1) * sizeof(float);

    /* Allocate device pointer for the matlab input on the gpu. */
    tcpda_data *d_thread_data;          /* TCPDA data struct for gpu memory */

	float *d_fbb;        				/* Bursts */
	float *d_fbg;
	float *d_fbr;
	float *d_fgg;
	float *d_fgr;
	
	int *d_NBGbb;       				/* NBGbb = numel(BGbb)-1 */
	int *d_NBGbg;
	int *d_NBGbr;
	int *d_NBGgg;
	int *d_NBGgr;
	
	float *d_BGbb;       				/* Background corrections */
	float *d_BGbg;
	float *d_BGbr;
	float *d_BGgg;
	float *d_BGgr;
	
	float *d_p_bb;       				/* percentage of 'PBB = Pout_B./P_total;' */
	float *d_p_bg;       
	float *d_p_gr;
    
    float *d_lib_b;
    float *d_lib_t;

    float *d_out_matrix_device;     	/* calculation space on the gpu */
    float *d_P_binomial;
    float *d_P_trinomial;
    	
	int *d_burst_cols;  				/* Dimension of Burst Array */
	int *d_burst_rows;
	
	int *d_p_cols;          			/* Dimension of P Array */  
	int *d_p_rows;

	int *d_thread_count;         		/* Number of threads running on gpu */
	int *d_block_count;

    /* Allocate the real memory on the gpu */
	/* tcpda struct */
    hipMalloc((void **) &d_thread_data, sizeof(tcpda_data));

	/* bursts */
    hipMalloc((void **) &d_fbb, size_burst_matrix);   
    hipMalloc((void **) &d_fbg, size_burst_matrix);   
    hipMalloc((void **) &d_fbr, size_burst_matrix);   
    hipMalloc((void **) &d_fgg, size_burst_matrix);   
    hipMalloc((void **) &d_fgr, size_burst_matrix);   

	/* NBG */
    hipMalloc((void **) &d_NBGbb, sizeof(int));   
    hipMalloc((void **) &d_NBGbg, sizeof(int));   
    hipMalloc((void **) &d_NBGbr, sizeof(int));   
    hipMalloc((void **) &d_NBGgg, sizeof(int));   
    hipMalloc((void **) &d_NBGgr, sizeof(int));   

	/* Background */
    hipMalloc((void **) &d_BGbb, size_burst_matrix);   
    hipMalloc((void **) &d_BGbg, size_burst_matrix);   
    hipMalloc((void **) &d_BGbr, size_burst_matrix);   
    hipMalloc((void **) &d_BGgg, size_burst_matrix);   
    hipMalloc((void **) &d_BGgr, size_burst_matrix); 

	/* P Matrix */
    hipMalloc((void **) &d_p_bb, size_p_matrix);   
    hipMalloc((void **) &d_p_bg, size_p_matrix);   
    hipMalloc((void **) &d_p_gr, size_p_matrix);   
    
    /* binomial and trinomial coefficient library */
    hipMalloc((void **) &d_lib_b, size_lib_b);
    hipMalloc((void **) &d_lib_t, size_lib_t);

    hipMalloc((void **) &d_out_matrix_device,(burst_rows * p_rows * sizeof(float)));   
    hipMalloc((void **) &d_P_binomial, (burst_rows * p_rows * sizeof(float)));   
    hipMalloc((void **) &d_P_trinomial, (burst_rows * p_rows * sizeof(float)));   
    hipMalloc((void **) &d_burst_cols, sizeof(int));   
    hipMalloc((void **) &d_burst_rows, sizeof(int)); 
    hipMalloc((void **) &d_p_cols, sizeof(int));   
    hipMalloc((void **) &d_p_rows, sizeof(int)); 

    hipMalloc((void **) &d_thread_count, sizeof(int));   
    hipMalloc((void **) &d_block_count, sizeof(int)); 


    /* Copy the content from the host to the device (gpu) and set calculation memory to zero. */
    hipMemset(d_out_matrix_device, 0, (burst_rows * p_rows * sizeof(float)));
    hipMemset(d_P_binomial, 0, (burst_rows * p_rows * sizeof(float)));
    hipMemset(d_P_trinomial, 0, (burst_rows * p_rows * sizeof(float)));

    hipMemcpy(d_fbb, ((float *)mxGetData(prhs[0])), size_burst_matrix, hipMemcpyHostToDevice);
    hipMemcpy(d_fbg, ((float *)mxGetData(prhs[1])), size_burst_matrix, hipMemcpyHostToDevice);
    hipMemcpy(d_fbr, ((float *)mxGetData(prhs[2])), size_burst_matrix, hipMemcpyHostToDevice);
    hipMemcpy(d_fgg, ((float *)mxGetData(prhs[3])), size_burst_matrix, hipMemcpyHostToDevice);
    hipMemcpy(d_fgr, ((float *)mxGetData(prhs[4])), size_burst_matrix, hipMemcpyHostToDevice);
    
    hipMemcpy(d_NBGbb, ((int *)mxGetData(prhs[5])), sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_NBGbg, ((int *)mxGetData(prhs[6])), sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_NBGbr, ((int *)mxGetData(prhs[7])), sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_NBGgg, ((int *)mxGetData(prhs[8])), sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_NBGgr, ((int *)mxGetData(prhs[9])), sizeof(int), hipMemcpyHostToDevice);
    
    hipMemcpy(d_BGbb, ((float *)mxGetData(prhs[10])), size_burst_matrix, hipMemcpyHostToDevice);
    hipMemcpy(d_BGbg, ((float *)mxGetData(prhs[11])), size_burst_matrix, hipMemcpyHostToDevice);
    hipMemcpy(d_BGbr, ((float *)mxGetData(prhs[12])), size_burst_matrix, hipMemcpyHostToDevice);
    hipMemcpy(d_BGgg, ((float *)mxGetData(prhs[13])), size_burst_matrix, hipMemcpyHostToDevice);
    hipMemcpy(d_BGgr, ((float *)mxGetData(prhs[14])), size_burst_matrix, hipMemcpyHostToDevice);
    
    hipMemcpy(d_p_bb, ((float *)mxGetData(prhs[15])), size_p_matrix, hipMemcpyHostToDevice);
    hipMemcpy(d_p_bg, ((float *)mxGetData(prhs[16])), size_p_matrix, hipMemcpyHostToDevice);
    hipMemcpy(d_p_gr, ((float *)mxGetData(prhs[17])), size_p_matrix, hipMemcpyHostToDevice);

    hipMemcpy(d_burst_cols, ((int *)&burst_cols), sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_burst_rows, ((int *)&burst_rows), sizeof(int), hipMemcpyHostToDevice);

    hipMemcpy(d_p_cols, ((int *)&p_cols), sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_p_rows, ((int *)&p_rows), sizeof(int), hipMemcpyHostToDevice);
    
    /* initialize lib_b and lib_t */
    /* size information requires background numbers */
    hipMemcpy(d_lib_b, ((float *)mxGetData(prhs[18])), size_lib_b, hipMemcpyHostToDevice);
    hipMemcpy(d_lib_t, ((float *)mxGetData(prhs[19])), size_lib_t, hipMemcpyHostToDevice);
    
    /* Update the device pointer to the real data. */
    hipMemcpy(&(d_thread_data->fbb), &d_fbb, sizeof(float*), hipMemcpyHostToDevice);
    hipMemcpy(&(d_thread_data->fbg), &d_fbg, sizeof(float*), hipMemcpyHostToDevice);
    hipMemcpy(&(d_thread_data->fbr), &d_fbr, sizeof(float*), hipMemcpyHostToDevice);
    hipMemcpy(&(d_thread_data->fgg), &d_fgg, sizeof(float*), hipMemcpyHostToDevice);
    hipMemcpy(&(d_thread_data->fgr), &d_fgr, sizeof(float*), hipMemcpyHostToDevice);

    hipMemcpy(&(d_thread_data->NBGbb), &d_NBGbb, sizeof(int*), hipMemcpyHostToDevice);
    hipMemcpy(&(d_thread_data->NBGbg), &d_NBGbg, sizeof(int*), hipMemcpyHostToDevice);
    hipMemcpy(&(d_thread_data->NBGbr), &d_NBGbr, sizeof(int*), hipMemcpyHostToDevice);
    hipMemcpy(&(d_thread_data->NBGgg), &d_NBGgg, sizeof(int*), hipMemcpyHostToDevice);
    hipMemcpy(&(d_thread_data->NBGgr), &d_NBGgr, sizeof(int*), hipMemcpyHostToDevice);

    hipMemcpy(&(d_thread_data->BGbb), &d_BGbb, sizeof(float*), hipMemcpyHostToDevice);
    hipMemcpy(&(d_thread_data->BGbg), &d_BGbg, sizeof(float*), hipMemcpyHostToDevice);
    hipMemcpy(&(d_thread_data->BGbr), &d_BGbr, sizeof(float*), hipMemcpyHostToDevice);
    hipMemcpy(&(d_thread_data->BGgg), &d_BGgg, sizeof(float*), hipMemcpyHostToDevice);
    hipMemcpy(&(d_thread_data->BGgr), &d_BGgr, sizeof(float*), hipMemcpyHostToDevice);

    hipMemcpy(&(d_thread_data->p_bb), &d_p_bb, sizeof(float*), hipMemcpyHostToDevice);
    hipMemcpy(&(d_thread_data->p_bg), &d_p_bg, sizeof(float*), hipMemcpyHostToDevice);
    hipMemcpy(&(d_thread_data->p_gr), &d_p_gr, sizeof(float*), hipMemcpyHostToDevice);    

    hipMemcpy(&(d_thread_data->burst_cols), &d_burst_cols, sizeof(int*), hipMemcpyHostToDevice);
    hipMemcpy(&(d_thread_data->burst_rows), &d_burst_rows, sizeof(int*), hipMemcpyHostToDevice);
    hipMemcpy(&(d_thread_data->p_cols), &d_p_cols, sizeof(int*), hipMemcpyHostToDevice);
    hipMemcpy(&(d_thread_data->p_rows), &d_p_rows, sizeof(int*), hipMemcpyHostToDevice);

    hipMemcpy(&(d_thread_data->lib_b), &d_lib_b, sizeof(float*), hipMemcpyHostToDevice);
    hipMemcpy(&(d_thread_data->lib_t), &d_lib_t, sizeof(float*), hipMemcpyHostToDevice);

    hipMemcpy(&(d_thread_data->out_matrix_device), &d_out_matrix_device, sizeof(float*), hipMemcpyHostToDevice);
    hipMemcpy(&(d_thread_data->P_binomial), &d_P_binomial, sizeof(float*), hipMemcpyHostToDevice);
    hipMemcpy(&(d_thread_data->P_trinomial), &d_P_trinomial, sizeof(float*), hipMemcpyHostToDevice);

    hipMemcpy(d_thread_count, ((int *)&thread_count), sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_block_count, ((int *)&block_count), sizeof(int), hipMemcpyHostToDevice);

    hipMemcpy(&(d_thread_data->thread_count), &d_thread_count, sizeof(int*), hipMemcpyHostToDevice);
    hipMemcpy(&(d_thread_data->block_count), &d_block_count, sizeof(int*), hipMemcpyHostToDevice);
    
    /* Start the calculation on the gpu device and check for errors */
    eval_prob_3c_bg <<< grid_dim,block_dim >>> (d_thread_data);
    hipPeekAtLastError();
    
    /* wait for all threads on the gpu to finish before copying the evaluated values back to the host */
    hipDeviceSynchronize();
    
    /* create 2d output matrix in Matlab for double precision (i.e. 2D double array) */
    plhs[0] = mxCreateNumericMatrix((mwSize)burst_rows,(mwSize)p_rows,mxSINGLE_CLASS, mxREAL);

    /* copy the calculated values back into matlab */
    hipMemcpy((float *)mxGetData(plhs[0]), d_out_matrix_device, (p_rows * burst_rows * sizeof(float)), hipMemcpyDeviceToHost);  

    /* Clean up *
    /* deallocate device memory on the gpu */
    hipFree(d_fbb);
    hipFree(d_fbg);
    hipFree(d_fbr);
    hipFree(d_fgg);
    hipFree(d_fgr);

    hipFree(d_NBGbb);
    hipFree(d_NBGbg);
    hipFree(d_NBGbr);
    hipFree(d_NBGgg);
    hipFree(d_NBGgr);

    hipFree(d_BGbb);
    hipFree(d_BGbg);
    hipFree(d_BGbr);
    hipFree(d_BGgg);
    hipFree(d_BGgr);

    hipFree(d_p_bb);
    hipFree(d_p_bg);
    hipFree(d_p_gr);

    hipFree(d_lib_b);
    hipFree(d_lib_t);

    hipFree(d_burst_cols);
    hipFree(d_burst_rows);
    hipFree(d_p_cols);
    hipFree(d_p_rows);

    hipFree(d_out_matrix_device);
    hipFree(d_P_binomial);
    hipFree(d_P_trinomial);

    hipFree(d_thread_count);
    hipFree(d_block_count);

    hipFree(d_thread_data);
}

 /* The computational routine */

__global__ void eval_prob_3c_bg(tcpda_data *thread_data)
{
    /* index is an identifier number of every single thread running on the gpu */
    int index = blockIdx.x * blockDim.x + threadIdx.x;
        
    /* Calculate the probability */

    /* Loop over all the probabilities */
    for (size_t i=0; i < *thread_data->p_rows; i++) {
        
        /* blue-red = blue-blue - blue-green */
        float p_br = 1 - thread_data->p_bb[i] - thread_data->p_bg[i];
        
        /* Loop over all bursts.
         * Here, we split up the work equally for the calculation threads
         * using the Thread ID in the data struct.
         * Every thread calculates the index+NUM_OF_TOTAL_THREADS'th 
         * probabilities, so we don't have to lock and synchronize the 
         * threads for writing the results, since the order does not
         * matter and no race conditions can occur.
         * For example: 6 (0-5) calculation threads are available, thread 
         * 0 calculates the probabilities for 0, 6, 12, 18, .. bursts.
         */
        for (size_t j=index; j < *thread_data->burst_rows; j=j+((*thread_data->thread_count) * (*thread_data->block_count))) {
            
            /* trinomal calculation */
            int bg_bb;
            for (size_t a=0; a <= MIN(thread_data->fbb[j], *thread_data->NBGbb) ; a++) {
                bg_bb = a;
                int bg_bg;
                for (size_t b=0; b <= MIN(thread_data->fbg[j], *thread_data->NBGbg) ; b++) {
                    bg_bg = b;
                    int bg_br;
                        for (size_t c=0; c <= MIN(thread_data->fbr[j], *thread_data->NBGbr) ; c++) {
                        bg_br = c;
                        
                        int ix = (int) j*(*thread_data->NBGbb+1)*(*thread_data->NBGbg+1)*(*thread_data->NBGbr+1)+bg_bb*(*thread_data->NBGbg+1)*(*thread_data->NBGbr+1)+bg_bg*(*thread_data->NBGbr+1)+bg_br;
                        
                        //float background[3] = {thread_data->p_bb[i], thread_data->p_bg[i], p_br};
                        //const unsigned int bursts[3] = {
                        //   (unsigned int) thread_data->fbb[j] - bg_bb,
                        //    (unsigned int) thread_data->fbg[j] - bg_bg,
                        //    (unsigned int) thread_data->fbr[j] - bg_br};

                        thread_data->P_trinomial[j+(*thread_data->burst_rows * i)] = thread_data->P_trinomial[j+(*thread_data->burst_rows * i)] +  // cumulative sum
                                thread_data->BGbb[bg_bb] *       // Background
                                thread_data->BGbg[bg_bg] *
                                thread_data->BGbr[bg_br] *
                                expf(
                                thread_data->lib_t[ix] +
                                logf(thread_data->p_bb[i])*(thread_data->fbb[j] - bg_bb) +
                                logf(thread_data->p_bg[i])*(thread_data->fbg[j] - bg_bg) +
                                logf(p_br)*(thread_data->fbr[j] - bg_br)
                                );
                                //ran_multinomial_pdf(3,background,bursts);
                        }
                }
            }

            /* bionomal calculation */

            size_t bg_gg;
            for (size_t a=0; a <= MIN(thread_data->fgg[j], *thread_data->NBGgg) ; a++) {
                bg_gg = a;
                size_t bg_gr;
                for (size_t b=0; b <= MIN(thread_data->fgr[j], *thread_data->NBGgr) ; b++) {
                    bg_gr = b;
                    
                    int ix = (int) j*(*thread_data->NBGgg+1)*(*thread_data->NBGgr+1)+bg_gg*(*thread_data->NBGgr+1)+bg_gr;
                    /* Subtract Background counts for FRET evaluation */
                    thread_data->P_binomial[j+(*thread_data->burst_rows * i)] = thread_data->P_binomial[j+(*thread_data->burst_rows * i)] +             // cumulative sum
                           thread_data->BGgg[bg_gg] *
                           thread_data->BGgr[bg_gr] *
                           expf(
                           thread_data->lib_b[ix] + 
                           logf(thread_data->p_gr[i])*(thread_data->fgr[j] - bg_gr) +
                           logf((1-thread_data->p_gr[i]))*(thread_data->fgg[j] - bg_gg)
                           );
                           //ran_binomial_pdf((unsigned int)(thread_data->fgr[j] - bg_gr),
                           //thread_data->p_gr[i], 
                           //(unsigned int) (thread_data->fgr[j] - bg_gr + thread_data->fgg[j] - bg_gg) );
                }
            }
        }
    }

    /* multiply element wise P_binomal by P_trinomal and fill the output array */
    for (size_t i=0; i < *thread_data->p_rows; i++) {
        for (size_t j=index; j < *thread_data->burst_rows; j=j+((*thread_data->thread_count) * (*thread_data->block_count))) {
            thread_data->out_matrix_device[j+(*thread_data->burst_rows * i)] = thread_data->P_binomial[j+(*thread_data->burst_rows * i)] * thread_data->P_trinomial[j+(*thread_data->burst_rows * i)];
        }
    }
        
    /* Clean up */
}

__device__
int sf_lnfact_e(const unsigned int n, sf_result * result)
{

  result->val = lgammaf(n+1.0);
  return EXIT_SUCCESS;

}

__device__
int sf_lnchoose_e(unsigned int n, unsigned int m, sf_result * result)
{
  /* CHECK_POINTER(result) */

  if(m > n) {
	  /* Handle Error */
    //DOMAIN_ERROR(result);
	return EXIT_FAILURE;
  }
  else if(m == n || m == 0) {
    result->val = 0.0;
    result->err = 0.0;
	return EXIT_SUCCESS;
  }
  else {
    sf_result nf;
    sf_result mf;
    sf_result nmmf;
    if(m*2 > n) m = n-m;
    sf_lnfact_e(n, &nf);
    sf_lnfact_e(m, &mf);
    sf_lnfact_e(n-m, &nmmf);
    result->val  = nf.val - mf.val - nmmf.val;
    //result->err  = nf.err + mf.err + nmmf.err;
    //result->err += 2.0 * GSL_DBL_EPSILON * fabs(result->val);
	return EXIT_SUCCESS;
  }
}

__device__
float sf_lnfact(const unsigned int n)
{
  EVAL_RESULT(sf_lnfact_e(n, &result));
}

__device__
float sf_lnchoose(unsigned int n, unsigned int m)
{
  EVAL_RESULT(sf_lnchoose_e(n, m, &result));
}

/* The multinomial distribution has the form

                                      N!           n_1  n_2      n_K
   prob(n_1, n_2, ... n_K) = -------------------- p_1  p_2  ... p_K
                             (n_1! n_2! ... n_K!) 

   where n_1, n_2, ... n_K are nonnegative integers, sum_{k=1,K} n_k = N,
   and p = (p_1, p_2, ..., p_K) is a probability distribution. 

   Random variates are generated using the conditional binomial method.
   This scales well with N and does not require a setup step.

   Ref: 
   C.S. David, The computer generation of multinomial random variates,
   Comp. Stat. Data Anal. 16 (1993) 205-217
*/
 
__device__
float
ran_multinomial_lnpdf (const size_t K,
                           const float p[], const unsigned int n[])
{
  size_t k;
  unsigned int N = 0;
  float log_pdf = 0.0;
  float norm = 0.0;

  for (k = 0; k < K; k++)
    {
      N += n[k];
    }

  for (k = 0; k < K; k++)
    {
      norm += p[k];
    }

  log_pdf = sf_lnfact (N);

  for (k = 0; k < K; k++)
    {
      /* Handle case where n[k]==0 and p[k]==0 */

      if (n[k] > 0) 
        {
          log_pdf += logf (p[k] / norm) * n[k] - sf_lnfact (n[k]);
        }
    }

  return log_pdf;
}

__device__
float
ran_multinomial_pdf (const size_t K,
                         const float p[], const unsigned int n[])
{
  return expf (ran_multinomial_lnpdf (K, p, n));
}

/* The binomial distribution has the form,

   prob(k) =  n!/(k!(n-k)!) *  p^k (1-p)^(n-k) for k = 0, 1, ..., n

   This is the algorithm from Knuth */

__device__
float
ran_binomial_pdf (const unsigned int k, const float p,
                      const unsigned int n)
{
  if (k > n)
    {
      return 0;
    }
  else
    {
      float P;

      if (p == 0) 
        {
          P = (k == 0) ? 1 : 0;
        }
      else if (p == 1)
        {
          P = (k == n) ? 1 : 0;
        }
      else
        {
          float ln_Cnk = sf_lnchoose (n, k);
          P = ln_Cnk + k * logf (p) + (n - k) * log1pf (-p);
          P = expf (P);
        }

      return P;
    }
}
