#include "hip/hip_runtime.h"
/* eval_prob_3c_bg.cu - Evaluate probabilty for 3-Color PDA after 
 *       blue and green excitation using CUDA
 *
 *========================================================*/

 /* Standard c libs */
#include <stdio.h>                       
#include <stdlib.h>
// #include <stddef.h>
// #include <limits.h>
#include <string.h>
#include <errno.h>

//#include "randist/randist_cuda.h"                        /* Statistical distributions */
#include "randist/sys/minmax.h"                            /* Minimum/Maximum Makro */

#include "randist/specfunc/eval_cuda.h"                    /* Data Structure of Calculations */
#include "randist/specfunc/result_cuda.h"                  /* Evaluvating results of calculations */

 /* For gpu calculation using CUDA */ 
#include <hip/hip_runtime.h>

 /* Forward declarations:
  * I couldn't split the source file without compile errors, even after customizing
  * mex_CUDA_win64.xml (contains matlab (mex) and cuda compile instructions for the
  * nvcc, the nvidia C/C++ Compiler) and adding '-rdc=true' as compile option and the
  * libraries 'cudart.lib' and 'cudadevrt.lib' */

__device__ int sf_lnfact_e(const unsigned int n, sf_result * result);
__device__ int sf_lnchoose_e(unsigned int n, unsigned int m, sf_result * result);
__device__ float sf_lnfact(const unsigned int n);
__device__ float sf_lnchoose(unsigned int n, unsigned int m);
__device__ float ran_multinomial_lnpdf (const size_t K, const float p[], const unsigned int n[]);
__device__ float ran_multinomial_pdf (const size_t K, const float p[], const unsigned int n[]);
__device__ float ran_binomial_pdf (const unsigned int k, const float p, const unsigned int n);

 /* The computational routine */

__global__ void eval_prob_3c_bg(
            float *likelihood,
 
            const int *fbb,
            const int *fbg,
            const int *fbr,
            const int *fgg,
            const int *fgr,

            const int NBGbb,
            const int NBGbg,
            const int NBGbr,
            const int NBGgg,
            const int NBGgr,

            const float *BGbb,
            const float *BGbg,
            const float *BGbr,
            const float *BGgg,
            const float *BGgr,

            const float *p_bb,
            const float *p_bg,       
            const float *p_gr,

            const int p_rows,
            const int n_bins            
)
{
    /* index is an identifier number of every single thread running on the gpu */
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    
    /* 
     * Every thread calculates one burst over all points of the grid
    */
    if (idx < n_bins){
        /* Calculate the probability */
        /* Loop over all the probabilities */
        for (int i=0; i < p_rows; i++) {

            /* blue-red = blue-blue - blue-green */        
            float prob[3] = {p_bb[i], p_bg[i], 1 - p_bb[i] - p_bg[i]};

            float P_binomial = 0;
            float P_trinomial = 0;

            /* trinomal calculation */
            int bg_bb;
            for (int a=0; a <= MIN(fbb[idx], NBGbb) ; a++) {
                bg_bb = a;
                int bg_bg;
                for (int b=0; b <= MIN(fbg[idx], NBGbg) ; b++) {
                    bg_bg = b;
                    int bg_br;
                    for (int c=0; c <= MIN(fbr[idx], NBGbr) ; c++) {
                        bg_br = c;
                        const unsigned int bursts[3] = {
                            (unsigned int) fbb[idx] - bg_bb,
                            (unsigned int) fbg[idx] - bg_bg,
                            (unsigned int) fbr[idx] - bg_br};

                        P_trinomial += BGbb[bg_bb] *       
                                BGbg[bg_bg] *
                                BGbr[bg_br] *         
                                ran_multinomial_pdf(3,prob,bursts);
                    }
                }
            }

            /* binomial calculation */
            int bg_gg;
            for (int a=0; a <= MIN(fgg[idx], NBGgg) ; a++) {
                bg_gg = a;
                int bg_gr;
                for (int b=0; b <= MIN(fgr[idx], NBGgr) ; b++) {
                   bg_gr = b;
                   /* Subtract Background counts for FRET evaluation */
                   P_binomial += BGgg[bg_gg] *
                           BGgr[bg_gr] *
                           ran_binomial_pdf((unsigned int)(fgr[idx] - bg_gr),
                           p_gr[i], 
                           (unsigned int) (fgr[idx] - bg_gr + fgg[idx] - bg_gg) );
                }
            }

            /* multiply both */
            likelihood[idx*p_rows + i] += P_trinomial*P_binomial;
        }
    }
}


__device__
int sf_lnfact_e(const unsigned int n, sf_result * result)
{

  result->val = lgammaf(n+1.0);
  return EXIT_SUCCESS;

}

__device__
int sf_lnchoose_e(unsigned int n, unsigned int m, sf_result * result)
{
  /* CHECK_POINTER(result) */

  if(m > n) {
	  /* Handle Error */
    //DOMAIN_ERROR(result);
	return EXIT_FAILURE;
  }
  else if(m == n || m == 0) {
    result->val = 0.0;
    result->err = 0.0;
	return EXIT_SUCCESS;
  }
  else {
    sf_result nf;
    sf_result mf;
    sf_result nmmf;
    if(m*2 > n) m = n-m;
    sf_lnfact_e(n, &nf);
    sf_lnfact_e(m, &mf);
    sf_lnfact_e(n-m, &nmmf);
    result->val  = nf.val - mf.val - nmmf.val;
    //result->err  = nf.err + mf.err + nmmf.err;
    //result->err += 2.0 * GSL_DBL_EPSILON * fabs(result->val);
	return EXIT_SUCCESS;
  }
}

__device__
float sf_lnfact(const unsigned int n)
{
  EVAL_RESULT(sf_lnfact_e(n, &result));
}

__device__
float sf_lnchoose(unsigned int n, unsigned int m)
{
  EVAL_RESULT(sf_lnchoose_e(n, m, &result));
}

/* The multinomial distribution has the form

                                      N!           n_1  n_2      n_K
   prob(n_1, n_2, ... n_K) = -------------------- p_1  p_2  ... p_K
                             (n_1! n_2! ... n_K!) 

   where n_1, n_2, ... n_K are nonnegative integers, sum_{k=1,K} n_k = N,
   and p = (p_1, p_2, ..., p_K) is a probability distribution. 

   Random variates are generated using the conditional binomial method.
   This scales well with N and does not require a setup step.

   Ref: 
   C.S. David, The computer generation of multinomial random variates,
   Comp. Stat. Data Anal. 16 (1993) 205-217
*/
 
__device__
float
ran_multinomial_lnpdf (const size_t K,
                           const float p[], const unsigned int n[])
{
  size_t k;
  unsigned int N = 0;
  float log_pdf = 0.0;
  float norm = 0.0;

  for (k = 0; k < K; k++)
    {
      N += n[k];
    }

  for (k = 0; k < K; k++)
    {
      norm += p[k];
    }

  log_pdf = sf_lnfact (N);

  for (k = 0; k < K; k++)
    {
      /* Handle case where n[k]==0 and p[k]==0 */

      if (n[k] > 0) 
        {
          log_pdf += logf (p[k] / norm) * n[k] - sf_lnfact (n[k]);
        }
    }

  return log_pdf;
}

__device__
float
ran_multinomial_pdf (const size_t K,
                         const float p[], const unsigned int n[])
{
  return expf (ran_multinomial_lnpdf (K, p, n));
}

/* The binomial distribution has the form,

   prob(k) =  n!/(k!(n-k)!) *  p^k (1-p)^(n-k) for k = 0, 1, ..., n

   This is the algorithm from Knuth */

__device__
float
ran_binomial_pdf (const unsigned int k, const float p,
                      const unsigned int n)
{
  if (k > n)
    {
      return 0;
    }
  else
    {
      float P;

      if (p == 0) 
        {
          P = (k == 0) ? 1 : 0;
        }
      else if (p == 1)
        {
          P = (k == n) ? 1 : 0;
        }
      else
        {
          float ln_Cnk = sf_lnchoose (n, k);
          P = ln_Cnk + k * logf (p) + (n - k) * log1pf (-p);
          P = expf (P);
        }

      return P;
    }
}
